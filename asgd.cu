#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <string.h>

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

#define SAMPLES 50
#define DIMENSIONALITY 10
#define THREADS 5
#define ITERATIONS 50
#define MINI_BATCH 10

#define STEP_SIZE 0.05

//read a csv of input data X, a matrix and store it
//implement function to calculate the gradient of of w wrt one datapoint
//implement update step
//implement kernel

int loadCSV(char *fileName, float *matrix, int rows, int cols);

__global__
void sgd_kernel(float* X, float* y, float *w,
                unsigned long long int num_samples,
                unsigned long long int dimensionality,
                unsigned long long int H,
                int num_iterations);

__device__
void take_gradient_step(float *X, float *w, float *y,
                        unsigned long long int threadId,
                        unsigned long long int t,
                        unsigned long long int dimensionality,
                        unsigned long long int num_samples );

int main()
{
    unsigned long long int num_samples = SAMPLES;
    unsigned long long int dimensionality = DIMENSIONALITY;
    int num_threads = THREADS;
    int num_iterations = ITERATIONS;

    unsigned long long int rows, cols;
    rows = num_samples;
    cols = dimensionality;

    // Seed RNG
    srand(time(NULL));

    // Allocate the arrays
    float *mat = (float*)malloc(rows * cols * sizeof(float));
    float *maty = (float*)malloc(rows * sizeof(float));
    
    int result = loadCSV((char*)"./xmatrix.csv", mat, rows, cols);
    int result2 = loadCSV((char*)"./yvector.csv", maty, 1, num_samples);

    if (result < 0 || result2 < 0)
    {
        printf("Unable to load file\n");
        return result;
    }

    int nThread = num_threads;
    unsigned long long int H = num_samples / nThread; //This is the number of samples each thread is going to handle
    if (num_samples % nThread != 0)
        nThread++;

    printf("H is %llu\n", H);

    float* X;
    float* w;
    float* y;
    float *host_w = (float*)malloc(nThread * dimensionality * sizeof(float));

    for (unsigned long long int i = 0; i < nThread; i++) {
        for (unsigned long long int j = 0; j < dimensionality; j++) {
            host_w[index(i, j, dimensionality)] = 0;
        }
    }

    hipError_t cudaStatus;
    cudaStatus = hipMalloc(&X, rows * cols * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc of X failed!");
        return -1;
    }

    cudaStatus = hipMalloc(&w, nThread * dimensionality * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc of w failed!");
        return -1;
    }

    cudaStatus = hipMalloc(&y, rows * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc of y failed!");
        return -1;
    }

    cudaStatus = hipMemcpy(X, mat, rows * cols * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy of X to device failed!");
        return -1;
    }

    cudaStatus = hipMemcpy(
        w, host_w, nThread * dimensionality * sizeof(float), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy of w to device failed!");
        return -1;
    }

    cudaStatus = hipMemcpy(y, maty, rows * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy of w to device failed!");
        return -1;
    }

    int blockWidth = 128;
    int totalBlocks = nThread / blockWidth;
    if (nThread % blockWidth != 0)
        totalBlocks++;

    printf("totalBLock is %d, blockWidth is %d\n", totalBlocks, blockWidth);
    sgd_kernel << <totalBlocks, blockWidth >> > (X, y, w, rows, cols, H, num_iterations);

    cudaStatus = hipMemcpy(
        host_w, w, nThread * dimensionality * sizeof(float), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy of w to host failed!");
        return -1;
    }

    printf("printing gpu result\n");

    for (unsigned long long int j = 0; j < dimensionality; j++)
    {
        for (int i = 0; i < nThread; i++)
        {
            printf("% f", host_w[index(i, j, dimensionality)]);
        }
        printf("\n");
    }
    printf("\n");

    //need to accumulate all the individual w vectors from each thread
    float *final_w = (float*)malloc(dimensionality * sizeof(float));

    for (int j = 0; j < dimensionality; j++)
    {
        float sum = 0;
        for (int i = 0; i < nThread; i++)
        {
            sum += host_w[index(i, j, dimensionality)];
        }
        final_w[j] = sum / nThread;
    }


    for (unsigned long long int j = 0; j < dimensionality; j++)
    {
        printf("%.6f ", final_w[j]);
    }
    printf("\n");

    free(mat);
    free(maty);
    free(host_w);
    free(final_w);

    return 0 ;

}

__global__
void sgd_kernel(float* X, float * y, float *w,
                unsigned long long int num_samples,
                unsigned long long int dimensionality,
                unsigned long long int H,
                int num_iterations)
{
    unsigned long long int s = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int startIndex = s * H;
    unsigned long long int endIndex = startIndex + H;

    /*if (s < 5){
        printf("%llu, %llu, %llu, %llu\n", num_samples, dimensionality, H, num_iterations);
        printf("thread id, %llu H is %llu, si, %llu ei %llu\n", s, H, startIndex, endIndex);
    }*/

    if (endIndex > num_samples)
        endIndex = num_samples;

    unsigned long long int guage = endIndex - startIndex;

    if (startIndex >= num_samples)
        return;

    //printf("thread id %llu, guage %llu\n", s, guage);

    for (int epoch = 0; epoch < num_iterations; epoch++)
    {
        for (int times = 0; times < guage; times++)
        {
            unsigned long long int t = startIndex + (times);
            //printf("thread id %llu, t %llu\n", s, t);
            //take a step with gradient on the t^{th} data point in X
            take_gradient_step(X, w, y, s, t, dimensionality, num_samples);
        }
    }
}

__device__
void take_gradient_step(float *X, float *w, float *y, 
                        unsigned long long int threadId,
                        unsigned long long int t,
                        unsigned long long int dimensionality,
                        unsigned long long int num_samples)
{
    float xwsum = 0;
    float term1[DIMENSIONALITY], term2[DIMENSIONALITY];
    float diff1, diff2;
    float temp = 0;
    int randomNode = (threadId == THREADS - 1) ? 0 : (threadId + 1);
    int delta;

    // Calculate X.W
    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        xwsum += X[index(t, i, dimensionality)] * w[index(threadId, i, dimensionality)];
    }
    xwsum -= y[t];
    
    // Build term1 and term 2
    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        term1[i] = w[index(threadId, i, dimensionality)] - 
                    STEP_SIZE * (xwsum * X[index(t, i, dimensionality)]);
    }

    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        term2[i] = w[index(randomNode, i, dimensionality)];
    }

    // Compute differences
    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        diff1 += term1[i] - term2[i];
    }

    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        diff2 += w[index(randomNode, i, dimensionality)] - w[index(threadId, i, dimensionality)];
    }

    // Compute delta
    diff1 = diff1 * diff1;
    diff2 = diff2 * diff2;
    delta = (diff1 < diff2) ? 1 : 0;

    // Compute [w^i - 1/2(w^i - w^j)] * delta
    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        temp += w[index(threadId, i, dimensionality)] - 
                w[index(threadId, randomNode, dimensionality)];
    }
    temp *= delta * 0.5;

    // Update weights
    for (unsigned long long int i = 0; i < dimensionality; i++)
    {
        w[index(threadId, i, dimensionality)] -= 
            STEP_SIZE * (temp + (xwsum * X[index(t, i, dimensionality)]));
    }
}

int loadCSV(char *fileName, float *matrix, int rows, int cols)
{
    char buffer[cols * 50];
    char *record, *line;
    int i = 0, j = 0;

    FILE *fstream = fopen(fileName, "r");
    if (fstream == NULL)
    {
        printf("\n file opening failed ");
        return -1 ;
    }
    while ((line = fgets(buffer, sizeof(buffer), fstream)) != NULL)
    {
        j = 0;
        record = strtok(line, ",");
        while (record != NULL)
        {
            matrix[index(i, j, cols)] = atof(record) ;
            j++;
            record = strtok(NULL, ",");
        }
        ++i ;
    }
    return 0;
}
